#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include <iostream>
#include <opencv2/cudev.hpp>
#include <>
#include <math.h>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/core/cuda/common.hpp>
#include "opencv2/imgcodecs.hpp"
#include "opencv2/cudaarithm.hpp"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include <stdlib.h>

using namespace std;
using namespace cv;

void BHEP(string path)
{
    Mat input, out;
    int frames = 0;
    VideoCapture cap(path);
    cap >> input;
    Mat gray(input.rows, input.cols, CV_8UC1);
    Mat output(input.rows, input.cols, CV_8UC1);
    int size = input.rows * input.cols;

    int b, g, r, hist[256], mean = 0, tl, tu, sl = 0, su = 0, f[256];
    float pdf[256], cdf[256];

    int frame_width = cap.get(CAP_PROP_FRAME_WIDTH);
    int frame_height = cap.get(CAP_PROP_FRAME_HEIGHT);
    VideoWriter outvid("BHEP.avi", VideoWriter::fourcc('M', 'J', 'P', 'G'), 30, Size(frame_width, frame_height), false);
    auto start = chrono::steady_clock::now();
    while (waitKey(27) != 27)
    {
        ++frames;
        cap >> input;
        if (input.empty())
            break;
        for (int i = 0; i < 256; i++)
            hist[i] = 0;
        for (int i = 0; i < input.rows; i++)
        {
            for (int j = 0; j < input.cols; j++)
            {
                b = input.at<Vec3b>(i, j)[0];
                g = input.at<Vec3b>(i, j)[1];
                r = input.at<Vec3b>(i, j)[2];
                gray.at<uchar>(i, j) = (b + g + r) / 3;
                hist[gray.at<uchar>(i, j)]++;
            }
        }
        cvtColor(input, input, COLOR_RGB2HLS);
        for (int i = 0; i < 256; i++)
            mean += i;
        mean = mean / 256;
        int sum = 0;
        for (int i = 0; i < 256; i++)
        {
            if (i <= mean)
                sl += hist[i];
            else
                su += hist[i];
            sum += hist[i];
        }
        tl = sl / (mean + 1);
        tu = su / (255 - mean);
        for (int i = 0; i < 256; i++)
        {
            if (i <= mean)
            {
                if (hist[i] > tl)
                    hist[i] = tl;
            }
            else
            {
                if (hist[i] > tu)
                    hist[i] = tu;
            }
        }
        float s1 = 0, s2 = 0;
        // waitKey(20);
        for (int i = 0; i < 256; i++)
        {
            if (i <= mean)
                s1 += hist[i];
            else
                s2 += hist[i];
        }
        for (int i = 0; i < 256; i++)
        {
            if ((i <= mean) & (s1 != 0))
                pdf[i] = hist[i] / s1;
            else if ((i > mean) & (s2 != 0))
                pdf[i] = hist[i] / s2;
            else
                pdf[i] = 0;
        }
        cdf[0] = pdf[0];
        cdf[mean + 1] = pdf[mean + 1];
        for (int i = 1; i < 256; i++)
        {
            if (i <= mean)
                cdf[i] = cdf[i - 1] + pdf[i];
            if (i > mean + 1)
                cdf[i] = cdf[i - 1] + pdf[i];
            //cout<<cdf[i]<<endl;
        }
        for (int i = 0; i < 256; i++)
        {
            if (i <= mean)
                f[i] = mean * (cdf[i] - 0.5 * pdf[i]);
            else
                f[i] = mean + 1 + (255 - (mean + 1)) * (cdf[i] - 0.5 * pdf[i]);
            //cout<<f[i]<<endl;
        }
        input.copyTo(out);
        cvtColor(out, out, COLOR_HLS2RGB);
        for (int i = 0; i < input.rows; i++)
        {
            for (int j = 0; j < input.cols; j++)
            {
                input.at<Vec3b>(i, j)[1] = f[gray.at<uchar>(i, j)];
            }
        }
        cvtColor(input, input, COLOR_HLS2RGB);
        // imshow("BHEP Output", input);
        // imshow("Input", out);
        outvid.write(output);
    }
    auto end = chrono::steady_clock::now();

    cout << "Elapsed time in nanoseconds: "
         << chrono::duration_cast<chrono::nanoseconds>(end - start).count()
         << " ns" << endl;

    cout << "Elapsed time in microseconds: "
         << chrono::duration_cast<chrono::microseconds>(end - start).count()
         << " µs" << endl;

    cout << "Elapsed time in milliseconds: "
         << chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << " ms" << endl;

    cout << "Elapsed time in seconds: "
         << chrono::duration_cast<chrono::seconds>(end - start).count()
         << " sec";
    cout << frames << endl;
}

__global__ void rgbtogray(const cv::cudev::PtrStepSz<uchar3> src, cv::cudev::PtrStepSz<uchar> dst, cv::cudev::PtrStepSz<int> histo)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < src.cols && y < src.rows)
    {
        dst(y, x) = (src(y, x).x + src(y, x).y + src(y, x).z) / 3;
        atomicAdd(&histo(dst(y, x), 0), 1);
    }
}

__global__ void calcLimits(cv::cudev::PtrStepSz<int> histo, float *sl, float *su)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x < 256)
    {
        if (x < 128)
            atomicAdd(sl, histo(0, x));
        else
            atomicAdd(su, histo(0, x));
    }
}

__global__ void thresh(cv::cudev::PtrStepSz<int> hist, float *tl, float *tu)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < 128)
    {
        if (hist(0, x) > int(*tl))
            hist(0, x) = *tl;
    }
    else
    {
        if (hist(0, x) > int(*tu))
            hist(0, x) = *tu;
    }
}

__global__ void calcPDF(cv::cudev::PtrStepSz<float> pdf, cv::cudev::PtrStepSz<int> hist, float *sl, float *su)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x < 128 & *sl != 0.0)
        pdf(0, x) = hist(0, x) / (*sl);
    else if (x > 127 & *su != 0.0)
        pdf(0, x) = hist(0, x) / (*su);
    else
        pdf(0, x) = 0;
}

__global__ void transformFunction(cv::cudev::PtrStepSz<float> pdf, cv::cudev::PtrStepSz<float> cdf)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x < 256)
    {
        if (x < 128)
            cdf(0, x) = 127 * (cdf(0, x) - 0.5 * pdf(0, x));
        else
            cdf(0, x) = 128 + 127 * (cdf(0, x) - 0.5 * pdf(0, x));
    }
}

__global__ void transformImage(cv::cudev::PtrStepSz<uchar> dst, cv::cudev::PtrStepSz<float> f)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < dst.cols && y < dst.rows)
    {
        dst(y, x) = f(dst(y, x), 0);
    }
}

int main()
{
    BHEP("input2n.mp4");
    cv::VideoCapture cap("input1n.mp4");
    cv::Mat img;
    cap.read(img);
    cv::Mat output(img.rows, img.cols, CV_8UC1);
    cv::Mat hist(256, 1, CV_32SC1), pdf(256, 1, CV_32FC1), cdf(256, 1, CV_32FC1);
    float *sl, *su, *tl, *tu;

    cv::cuda::GpuMat src;
    cv::cuda::GpuMat dst(img.rows, img.cols, CV_8UC1), gpu_hist(256, 1, CV_32SC1), gpu_pdf(256, 1, CV_32FC1), gpu_cdf(256, 1, CV_32FC1);
    float *gpu_f1, *gpu_f2;

    const dim3 block(16, 16);
    const dim3 grid(cv::cudev::divUp(img.cols, block.x), cv::cudev::divUp(img.rows, block.y));

    sl = (float *)malloc(sizeof(int));
    su = (float *)malloc(sizeof(int));
    tl = (float *)malloc(sizeof(int));
    tu = (float *)malloc(sizeof(int));

    hipMalloc((void **)&gpu_f1, sizeof(float));
    hipMalloc((void **)&gpu_f2, sizeof(float));
    auto start = chrono::steady_clock::now();
    int f = 0;

    while (cv::waitKey(40) != 27)
    {
        f++;
        cap >> img;

        if (img.cols == 0 || img.rows == 0)
        {
            break;
        }

        for (int i = 0; i < 256; i++)
        {
            // std::cout << pdf.at<float>(i, 0) << std::endl;
            cdf.at<float>(i, 0) = 0;
            pdf.at<float>(i, 0) = 0;
            hist.at<float>(i, 0) = 0;
        }

        *sl = 0;
        *su = 0;
        *tl = 0;
        *tu = 0;

        hipMemcpy(gpu_f1, sl, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(gpu_f2, su, sizeof(float), hipMemcpyHostToDevice);

        src.upload(img);

        rgbtogray<<<grid, block>>>(src, dst, gpu_hist);

        dst.download(output);

        // cv::imshow("input", output);

        gpu_hist.download(hist);

        calcLimits<<<1, 256>>>(gpu_hist, gpu_f1, gpu_f2);

        hipMemcpy(sl, gpu_f1, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(su, gpu_f2, sizeof(float), hipMemcpyDeviceToHost);

        *tl = *sl / 128;
        *tu = *su / 127;

        hipMemcpy(gpu_f1, tl, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(gpu_f2, tu, sizeof(float), hipMemcpyHostToDevice);

        thresh<<<1, 256>>>(gpu_hist, gpu_f1, gpu_f2);

        *sl = 0.0;
        *su = 0.0;
        hipMemcpy(gpu_f1, sl, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(gpu_f2, su, sizeof(float), hipMemcpyHostToDevice);

        calcLimits<<<1, 256>>>(gpu_hist, gpu_f1, gpu_f2);

        hipMemcpy(sl, gpu_f1, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(su, gpu_f2, sizeof(float), hipMemcpyDeviceToHost);

        calcPDF<<<1, 256>>>(gpu_pdf, gpu_hist, gpu_f1, gpu_f2);

        gpu_pdf.download(pdf);

        cdf.at<float>(0, 0) = pdf.at<float>(0, 0);
        cdf.at<float>(127, 0) = pdf.at<float>(127, 0);

        for (int i = 1; i < 256; i++)
        {
            if (i < 127)
                cdf.at<float>(i, 0) = cdf.at<float>(i - 1, 0) + pdf.at<float>(i, 0);
            if (i > 127)
                cdf.at<float>(i, 0) = cdf.at<float>(i - 1, 0) + pdf.at<float>(i, 0);
            // std::cout << cdf.at<float>(i - 1, 0) << std::endl;
        }

        gpu_cdf.upload(cdf);

        transformFunction<<<1, 256>>>(gpu_pdf, gpu_cdf);

        transformImage<<<grid, block>>>(dst, gpu_cdf);

        dst.download(output);

        // cv::imshow("Output", output);
    }
    auto end = chrono::steady_clock::now();

    cout << f << " Elapsed time in nanoseconds: "
         << chrono::duration_cast<chrono::nanoseconds>(end - start).count()
         << " ns" << endl;

    cout << "Elapsed time in microseconds: "
         << chrono::duration_cast<chrono::microseconds>(end - start).count()
         << " µs" << endl;

    cout << "Elapsed time in milliseconds: "
         << chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << " ms" << endl;

    cout << "Elapsed time in seconds: "
         << chrono::duration_cast<chrono::seconds>(end - start).count()
         << " sec";

    // cv::waitKey(0);

    return 0;
}