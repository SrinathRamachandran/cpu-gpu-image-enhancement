#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/cudev.hpp>
#include <>
#include "opencv2/opencv.hpp"

__global__ void rgbtogray(const cv::cudev::PtrStepSz<uchar3> src, cv::cudev::PtrStepSz<uchar> dst)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(x < src.cols && y < src.rows)
    {
        dst(y, x) = (0.114*src(y, x).x + 0.587*src(y, x).y + 0.299*src(y, x).z);
    }
}

int main (int argc, char* argv[])
{
    cv::Mat img = cv::imread("/home/srinath/FYP/tig.jpg", 1);
    cv::cuda::GpuMat src;
    cv::cuda::GpuMat dst(img.rows, img.cols, CV_8UC1);
    src.upload(img);

    // const dim3 block(64, 2);
	// const dim3 grid(cv::cudev::divUp(img.cols, block.x), cv::cudev::divUp(img.rows, block.y));
    const dim3 block(16,16);

	// Calculate grid size to cover the whole image
	const dim3 grid(cv::cudev::divUp(img.cols, block.x), cv::cudev::divUp(img.rows, block.y));
    std::cout << std::endl << grid.x << "  " << grid.y << std::endl;
    rgbtogray<<<grid, block>>>(src, dst);

    CV_CUDEV_SAFE_CALL(hipGetLastError());
	CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());

    cv::Mat result;
    dst.download(result);

    cv::imshow("result", result);
    cv::waitKey();
    
}